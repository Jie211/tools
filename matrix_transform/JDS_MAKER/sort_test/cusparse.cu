#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "mmio.h"
/* #include <hipsparse.h> */

#define CLEANUP(s)  \
do{                 \
  printf("%s\n", s);\
  if(f) fclose(f);\
  if(cooRow) free(cooRow);\
  if(cooCol) free(cooCol);\
  if(cooVal) free(cooVal);\
  if(matrix) free(matrix);\
  if(csrptr)  free(csrptr);\
  if(d_cooCol) hipFree(d_cooCol);\
  if(d_cooRow) hipFree(d_cooRow);\
  if(d_cooVal) hipFree(d_cooVal);\
  if(descr) hipsparseDestroyMatDescr(descr);\
  if(handle) hipsparseDestroy(handle);\
  if(output1) fclose(output1);\
  if(output2) fclose(output2);\
  if(output3) fclose(output3);\
  if(x) free(x);\
  if(b) free(b);\
  hipDeviceReset();        \
  fflush(stdout);           \
}while(0)

typedef struct{
  int row;
  int col;
  double val;
}mat;

int comp(const void *x, const void *y)
{
  if(x == y) return 0;
  if(x == 0) return -1;
  if(y == 0) return 1;

  const mat *ptr1 = (const mat*)x, *ptr2 = (const mat*)y;

  if(ptr1->row < ptr2->row) return -1;
  if(ptr1->row > ptr2->row) return 1;

  if(ptr1->col < ptr2->col) return -1;
  if(ptr1->col > ptr2->col) return 1;

  return 0;
}

int main(int argc, char *argv[]){
  srand((unsigned)time(NULL));
  int *cooRow=0;
  int *cooCol=0;
  double *cooVal=0;
  mat *matrix;
  
  int *d_cooRow = 0;
  int *d_cooCol = 0;
  double *d_cooVal = 0;
  
  int N, M;
  int NNZ;

  hipError_t stat1, stat2, stat3;
  hipsparseHandle_t handle = 0;
  hipsparseMatDescr_t descr = 0;

  int *d_csrptr = 0;
  int *csrptr = 0;

  int size_read;
  MM_typecode type_read;
  FILE *f;

  FILE *output1;
  FILE *output2;

  double getMAX = 0;
  double getMIN = 0;

  FILE *output3;
  double *x = 0;
  double *b = 0;
  double xvm;

if(argc<2)
  {
    fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
    exit(0);
  }else
  {
    if((f = fopen(argv[1], "r")) == NULL)
      exit(1);
  }

  if(mm_read_banner(f, &type_read)!= 0)
  {
    printf("could not process matrix market banner.\n");
    exit(1);
  }

  if(mm_is_complex(type_read) && mm_is_matrix(type_read) && mm_is_sparse(type_read))
  {
    printf("this app does not support ");
    printf("Market market type '[%s]\n", mm_typecode_to_str(type_read));
  }

  if((size_read = mm_read_mtx_crd_size(f, &M, &N, &NNZ))!= 0)
    exit(1);

  printf("~~~~~~~~~~~~~~~~~~~~~~~~~\n");
  mm_write_mtx_crd_size(stdout, M, N, NNZ);
  printf("~~~~~~~~~~~~~~~~~~~~~~~~~\n");

  if(M!=N){
    CLEANUP("M!=N size error");
    return  -1;
  }
  printf("N=%d NNZ=%d\n",N,NNZ);
  int NNZ_old;
  NNZ_old = NNZ;
  NNZ = NNZ*2 - N;
  printf("NNZ %d was fix to %d\n", NNZ_old, NNZ);
  cooRow=(int *)malloc(sizeof(cooRow[0])*NNZ);
  cooCol=(int *)malloc(sizeof(cooCol[0])*NNZ);
  cooVal=(double *)malloc(sizeof(cooVal[0])*NNZ);
  matrix = (mat *)malloc(sizeof(mat)*NNZ);


  if((!cooRow)||(!cooCol)||(!cooVal)){
    CLEANUP("Host malloc matrix coo failed");
    return  -1;
  }
  if((!matrix)){
    CLEANUP("Host malloc matrix failed");
    exit(1);
  }
  int r_i;
  int r_j;
  double r_val;
  printf("Input data_______\n");
  for(int i = 0;i<NNZ_old;i++)
  {
    fscanf(f, "%d %d %lg\n", &r_i, &r_j, &r_val);
    /* cooRow[i] = r_i - 1; */
    /* cooCol[i] = r_j - 1; */
    /* cooVal[i] = r_val; */
    matrix[i].row = r_i - 1;
    matrix[i].col = r_j - 1;
    matrix[i].val = r_val;
  }
  int NNZcount = 0;
  for(int i = 0;i<NNZ_old;i++)
  {
    if(matrix[i].row != matrix[i].col)
    {
      /* cooRow[NNZ_old+NNZcount] = cooCol[i]; */
      /* cooCol[NNZ_old+NNZcount] = cooRow[i]; */
      /* cooVal[NNZ_old+NNZcount] = cooVal[i]; */
      matrix[NNZ_old+NNZcount].row = matrix[i].col;
      matrix[NNZ_old+NNZcount].col = matrix[i].row;
      matrix[NNZ_old+NNZcount].val = matrix[i].val;
      NNZcount++;
    }
  }


  /* for(int i = 0;i<NNZ;i++){ */
  /*   printf("index[%d]->", i); */
  /*   printf("Row=%d ", matrix[i].row); */
  /*   printf("Col=%d ", matrix[i].col); */
  /*   printf("Val=%f\n", matrix[i].val); */
  /* } */
  printf("Input over\n");
  printf("start sort_________\n");

  /* sort(cooRow, cooCol, cooVal, NNZ); */
  qsort(matrix, NNZ, sizeof(mat), comp);
  /* for(int i = 0;i<NNZ;i++){ */
  /*   printf("index[%d]->", i); */
  /*   printf("Row=%d ", cooRow[i]); */
  /*   printf("Col=%d ", cooCol[i]); */
  /*   printf("Val=%f\n", cooVal[i]); */
  /* } */
  printf("sort over\n");
  printf("getMAX getMIN\n");
  getMAX = getMIN = cooVal[0];
  for(int i = 0;i<NNZ;i++)
  {
    /* if(cooVal[i]>=getMAX){ */
    if(matrix[i].val>=getMAX){
      /* getMAX = cooVal[i]; */
      getMAX = matrix[i].val;
    }
    /* if(cooVal[i]<=getMIN){ */
    if(matrix[i].val<=getMIN){
      /* getMIN = cooVal[i]; */
      getMIN = matrix[i].val;
    }
  }
 
  for(int i = 0;i<NNZ;i++)
  {
    cooRow[i] = matrix[i].row;
    cooCol[i] = matrix[i].col;
    cooVal[i] = matrix[i].val;
  }

  printf("MAX = %f\nMIN = %f\n", getMAX, getMIN);
  printf("coo device malloc\n");
  stat1 = hipMalloc((void**)&d_cooRow, sizeof(d_cooRow)*NNZ);
  stat2 = hipMalloc((void**)&d_cooCol, sizeof(d_cooCol)*NNZ);
  stat3 = hipMalloc((void**)&d_cooVal, sizeof(d_cooVal)*NNZ);
  if((stat1!= hipSuccess)||
      (stat2!= hipSuccess)||
      (stat3!= hipSuccess)){
    CLEANUP("Device malloc failed");
    return  -1;
  }
  printf("memcpy coo to device\n");
  stat1 = hipMemcpy(d_cooRow, cooRow, (size_t)(sizeof(d_cooRow[0])*NNZ), hipMemcpyHostToDevice);
  stat2 = hipMemcpy(d_cooCol, cooCol, (size_t)(sizeof(d_cooCol[0])*NNZ), hipMemcpyHostToDevice);
  stat3 = hipMemcpy(d_cooVal, cooVal, (size_t)(sizeof(d_cooVal[0])*NNZ), hipMemcpyHostToDevice);
  if((stat1!= hipSuccess)||
      (stat2!= hipSuccess)||
      (stat3!= hipSuccess)){
    CLEANUP("Memcpy host 2 device failed");
    return  -1;
  }
  hipsparseStatus_t custat;
  printf("create handle and descriptor\n"); 
  custat = hipsparseCreate(&handle);
  if(custat!= HIPSPARSE_STATUS_SUCCESS){
    CLEANUP("CUSPARSE lib initialization failed");
    return 1;
  }
  custat = hipsparseCreateMatDescr(&descr);
  if(custat!= HIPSPARSE_STATUS_SUCCESS){
    CLEANUP("Matrix descriptor initialization failed");
    return 1;
  }
  printf("set descriptor type and indexbase\n");
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  csrptr = (int *)malloc(sizeof(csrptr[0])*(N + 1));
  if(!csrptr){
    CLEANUP("csrptr malloc failed");
    return  -1;
  }
  stat1 =  hipMalloc((void**)&d_csrptr, sizeof(d_csrptr[0])*(N+1));
  if(stat1!=hipSuccess){
    CLEANUP("Device malloc failed");
    return 1;
  }
  printf("conversion ptr vec___________\n");
  custat = hipsparseXcoo2csr(handle, d_cooRow, NNZ, N, d_csrptr, HIPSPARSE_INDEX_BASE_ZERO);
  if(custat!=HIPSPARSE_STATUS_SUCCESS){
    CLEANUP("conversion coo 2 csr failed");
    return 1;
  }
  stat1 = hipMemcpy(csrptr, d_csrptr, (size_t)(sizeof(d_csrptr[0])*(N+1)), hipMemcpyDeviceToHost);
  if(stat1!=hipSuccess){
    CLEANUP("Memcpy d_csrptr  ->csrptr failed ");
    return 1;
  }
  printf("conversion done\n");
  /* for(int i = 0;i<N +1;i++){ */
  /*   printf("csrptr[%d]=%d\n", i, csrptr[i]); */
  /* } */
  printf("create x random by %f~%f\n", getMIN, getMAX);
  x = (double *)malloc(sizeof(x[0])*N);
  b = (double *)malloc(sizeof(b[0])*N);
  for(int i = 0;i<N;i++)
  {
    /* if(fabs(getMAX)>=RAND_MAX || fabs(getMIN)>=RAND_MAX) */
    /* { */
    /*   x[i] = rand(); */
    /* }else{ */
    /*   x[i] = (double)(rand()%((int)getMAX*10 - (int)getMIN*10 + 1)+(int)getMIN*10)/10.0; */
    /* } */
    x[i] = 1.0;
  }
  printf("create b by x and Matrix\n");
  for(int i = 0;i<N;i++)
  {
    xvm = 0.0;
    for(int j = csrptr[i];j<csrptr[i+1];j++)
    {
      xvm+=cooVal[j]*x[cooCol[j]];
    }
    b[i] = xvm;
  }
  printf("ready to output________\n");
  if((output1 = fopen("./ColVal.txt", "w")) == NULL)
      exit(1); 
  if((output2 = fopen("./Ptr.txt", "w")) == NULL)
      exit(1);
  if((output3 = fopen("./bx.txt", "w")) == NULL)
      exit(1); 
  
  printf("output col val\n");
  fprintf(output1, "%d %d %d\n", N, M, NNZ);
  for(int i = 0;i<NNZ;i++)
  {
    fprintf(output1, "%d %d %.16e\n", cooRow[i], cooCol[i], cooVal[i]);
  }
  printf("output ptr\n");
  fprintf(output2, "%d %d %d\n", N, M, NNZ);
  for(int i = 0;i<N+1; i++)
  {
    fprintf(output2, "%d\n", csrptr[i]);
  }
  printf("output b x\n");
  fprintf(output3, "%d %d %d\n", N, M, NNZ);
  for(int i = 0;i<N;i++)
  {
    fprintf(output3, "%.16e %.16e\n", b[i], x[i]);
  }

  printf("output over \n");

  CLEANUP("Program safety end\n");
  return 0;
}
