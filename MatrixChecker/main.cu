#include "header.h"

#define CLEANUP(s) \
  do{               \
    printf("%s\n",s);\
    if(b) free(b);\
    if(x) free(x);\
    if(jd) free(jd);\
    if(perm) free(perm);\
    if(col) free(col);\
    if(nz) free(nz);\
    if(val) free(val);\
    hipDeviceReset();\
    fflush(stdout);\
  }while(0)\

int main(int argc, char const* argv[])
{
  int N;
  int M;
  int MAXNZ;

  double *b;
  double *x;
  int *jd;
  int *perm;
  int *col;
  int *nz;
  double *val;
  
  char copy[256];
  const char *hoge;
  const char *hoge2;
  const char *hoge3;

  if(argc!=5)
  {
    printf("Usage error\n");
    printf("%s mat_col perm_nz jd  bx \n", argv[0]);
    exit(0);
  }

#ifdef EBUG
  printf("@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n");
  printf("get headpart and check headerror\n");
#endif
  gethead_jad(argv[1],argv[2],argv[3],argv[4], &N,&M); 
#ifdef EBUG
  printf("get N = %d M = %d\n", N, M);
  printf("~~~~~~~ header ok ~~~~~~~~\n");
#endif

  strcpy(copy, argv[1]);
  hoge = strtok(copy, "/");
  hoge2 = strtok(NULL, "/");
  hoge3 = strtok(NULL, "/");
  printf("Matrix = %s\n", hoge3);
  strcpy(copy, hoge2);
  strcpy(copy, hoge);

  b=(double *)malloc(sizeof(double)*N);
  x=(double *)malloc(sizeof(double)*N);

  if(!b || !x)
  {
    CLEANUP("malloc b x error");
  }

  getmaxnz_jad(argv[2],&MAXNZ);
  val=(double *)malloc(sizeof(double)*M);
  col=(int *)malloc(sizeof(int)*M);
  perm = (int *)malloc(sizeof(int)*N);
  nz = (int *)malloc(sizeof(int)*N);
  jd = (int *)malloc(sizeof(int)*(MAXNZ+1));

  if(!val || !col || !perm || !nz || !jd)
  {
    CLEANUP("malloc val col perm nz jd");
  }

#ifdef EBUG  
  printf("read data\n");
#endif
 getdata_jad(argv[1], argv[2], argv[3], argv[4], col, perm, nz, jd, val, b, x, N, M, MAXNZ);

#ifdef EBUG
  printf("read data over\n");
  printf("@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n");
#endif

  printf("Max = %d, Min = %d\n", nz[0], nz[N - 1]);
  int nz_sum = 0;
  for(int i = 0;i<N-1;i++){
    /* printf("%d %d\n", i, nz[i]); */
    nz_sum+=nz[i];
  }
  printf("Avg = %3.1f\n", (double)nz_sum/(double)N);

   free(b); 
   free(x); 
   free(val);
   free(col);
   free(perm); 
   free(nz);
   free(jd);

  return 0;
}
